#include "hip/hip_runtime.h"


#include "utils.h"
#include "magma_types.h"
#include "operation_batched.h"
#include <hip/hip_runtime.h>
#include ""


/*
    -- MAGMA (version 2.5.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date August 2019

       @author Tingxing Dong
       @author Azzam Haidar

       @generated from magmablas/ztrsv_batched.cu, normal z -> s, Fri Aug  2 17:10:10 2019
*/


#define PRECISION_s

#define NB 256  //NB is the 1st level blocking in recursive blocking, BLOCK_SIZE is the 2ed level, NB=256, BLOCK_SIZE=64 is optimal for batched

#define NUM_THREADS 128 //64 //128

#define BLOCK_SIZE_N 128
#define DIM_X_N 128
#define DIM_Y_N 1

#define BLOCK_SIZE_T 32
#define DIM_X_T 16
#define DIM_Y_T 8


#define A(i, j)  (A + (i) + (j)*lda)   // A(i, j) means at i row, j column

extern __shared__ float shared_data[];


template<magma_trans_t transA, magma_diag_t diag>
static __device__ void
strsv_backwards_tri_device(int n,
    const float* __restrict__ A, int lda,
    float* __restrict__ b, int incb,
    float* sx)

{
    /*
    assume sx is in shared memory
    */
    int tx = threadIdx.x;
    float a;

    for (int step = 0; step < n; step++)
    {
        if (tx < n)
        {
            if (transA == MagmaNoTrans)
            {
                a = A[(n - 1) + (n - 1) * lda - tx - step * lda]; // rowwise access data in a coalesced way
            }
            else if (transA == MagmaTrans)
            {
                a = A[(n - 1) + (n - 1) * lda - tx * lda - step]; // columwise access data, not in a coalesced way
            }
            else
            {
                a = MAGMA_S_CONJ(A[(n - 1) + (n - 1) * lda - tx * lda - step]); // columwise access data, not in a coalesced way
            }


            if (tx == step)
            {
                if (diag == MagmaUnit)
                {
                    sx[n - 1 - tx] = (b[n - 1 - tx] - sx[n - 1 - tx]);
                }
                else
                {
                    sx[n - 1 - tx] = (b[n - 1 - tx] - sx[n - 1 - tx]) / a;
                }
            }
        }
        __syncthreads(); // there should be a sych here but can be avoided if BLOCK_SIZE =32

        if (tx < n)
        {
            if (tx > step)
            {
                sx[n - 1 - tx] += a * sx[n - 1 - step];
            }
        }
    }
}


#define make_FloatingPoint(x, y) (x)

template< int n, typename T >
__device__ void
magma_sum_reduce( /*int n,*/ int i, T* x)
{
    __syncthreads();
    if (n > 1024) { if (i < 1024 && i + 1024 < n) { x[i] += x[i + 1024]; }  __syncthreads(); }
    if (n > 512) { if (i < 512 && i + 512 < n) { x[i] += x[i + 512]; }  __syncthreads(); }
    if (n > 256) { if (i < 256 && i + 256 < n) { x[i] += x[i + 256]; }  __syncthreads(); }
    if (n > 128) { if (i < 128 && i + 128 < n) { x[i] += x[i + 128]; }  __syncthreads(); }
    if (n > 64) { if (i < 64 && i + 64 < n) { x[i] += x[i + 64]; }  __syncthreads(); }
    if (n > 32) { if (i < 32 && i + 32 < n) { x[i] += x[i + 32]; }  __syncthreads(); }
    // probably don't need __syncthreads for < 16 threads
    // because of implicit warp level synchronization.
    if (n > 16) { if (i < 16 && i + 16 < n) { x[i] += x[i + 16]; }  __syncthreads(); }
    if (n > 8) { if (i < 8 && i + 8 < n) { x[i] += x[i + 8]; }  __syncthreads(); }
    if (n > 4) { if (i < 4 && i + 4 < n) { x[i] += x[i + 4]; }  __syncthreads(); }
    if (n > 2) { if (i < 2 && i + 2 < n) { x[i] += x[i + 2]; }  __syncthreads(); }
    if (n > 1) { if (i < 1 && i + 1 < n) { x[i] += x[i + 1]; }  __syncthreads(); }
}
// end sum_reduce

template<typename T, const int DIM_X, const int DIM_Y, const int TILE_SIZE>
static __device__ void
gemvn_template_device(
    int m, int n, T alpha,
    const T* __restrict__ A, int lda,
    const T* __restrict__ x, int incx, T beta,
    T* __restrict__ y, int incy)
{
    if (m <= 0 || n <= 0) return;

    int num_threads = blockDim.x * blockDim.y * blockDim.z;

    if (DIM_X * DIM_Y != num_threads) return; // need to launch exactly the same number of threads as template parameters indicate

    int thread_id = threadIdx.x + threadIdx.y * blockDim.x;

    // threads are all configurated locally
    int tx = thread_id % DIM_X;
    int ty = thread_id / DIM_X;

    int ind = blockIdx.x * TILE_SIZE + tx;

    __shared__ T sdata[DIM_X * DIM_Y];


    int st = blockIdx.x * TILE_SIZE;

    int ed = min(st + TILE_SIZE, magma_roundup(m, DIM_X));

    int iters = (ed - st) / DIM_X;

    for (int i = 0; i < iters; i++)
    {
        if (ind < m) A += ind;

        T res = make_FloatingPoint(0.0, 0.0);

        if (ind < m)
        {
            for (int col = ty; col < n; col += DIM_Y)
            {
                res += A[col * lda] * x[col * incx];
            }
        }

        if (DIM_X >= num_threads) // indicated 1D threads configuration. Shared memory is not needed, reduction is done naturally
        {
            if (ty == 0 && ind < m)
            {
                y[ind * incy] = alpha * res + beta * y[ind * incy];
            }
        }
        else
        {
            sdata[ty + tx * DIM_Y] = res;

            __syncthreads();

            if (DIM_Y > 16)
            {
                magma_sum_reduce< DIM_Y >(ty, sdata + tx * DIM_Y);
            }
            else
            {
                if (ty == 0 && ind < m)
                {
                    for (int i = 1; i < DIM_Y; i++)
                    {
                        sdata[tx * DIM_Y] += sdata[i + tx * DIM_Y];
                    }
                }
            }

            if (ty == 0 && ind < m)
            {
                y[ind * incy] = alpha * sdata[tx * DIM_Y] + beta * y[ind * incy];
            }

            __syncthreads();
        }

        if (ind < m) A -= ind;

        ind += DIM_X;
    }
}


/******************************************************************************/
/*
    used in lower nontranspose and upper transpose
*/
template<magma_trans_t transA, magma_diag_t diag>
static __device__ void
strsv_forwards_tri_device(int n,
    const float* __restrict__ A, int lda,
    float* __restrict__ b, int incb,
    float* sx)

{
    /*
    assume sx is in shared memory
    */
    int tx = threadIdx.x;
    float a;

    for (int step = 0; step < n; step++)
    {
        if (tx < n) // hard code to BLOCK_SIZE and test divisible case only make 1Gflop/s difference
        {
            if (transA == MagmaNoTrans)
            {
                a = A[tx + step * lda]; // rowwise access data in a coalesced way
            }
            else  if (transA == MagmaTrans)
            {
                a = A[tx * lda + step]; // columwise access data, not in a coalesced way
            }
            else
            {
                a = MAGMA_S_CONJ(A[tx * lda + step]); // columwise access data, not in a coalesced way
            }


            if (tx == step)
            {
                if (diag == MagmaUnit)
                {
                    sx[tx] = (b[tx] - sx[tx]);
                }
                else
                {
                    sx[tx] = (b[tx] - sx[tx]) / a;
                }
            }
        }
        __syncthreads(); // there should be a sych here but can be avoided if BLOCK_SIZE =32

        if (tx < n)
        {
            if (tx > step)
            {
                sx[tx] += a * sx[step];
            }
        }
    }
}

template<const int BLOCK_SIZE, const int BLK_X, const int BLK_Y, const int TILE_SIZE, const int flag, const magma_uplo_t uplo, const magma_trans_t trans, const magma_diag_t diag>
static __device__ void
strsv_notrans_device(
    int n,
    const float* __restrict__ A, int lda,
    float* b, int incb,
    float* x)
{
    int tx = threadIdx.x;
    int col = n;
    float* sx = (float*)shared_data;

    if (flag == 0)
    {
        for (int j = tx; j < n; j += BLOCK_SIZE)
        {
            sx[j] = MAGMA_S_ZERO;
        }
    }
    else
    {
        for (int j = tx; j < n; j += BLOCK_SIZE)
        {
            sx[j] = x[j];
        }
    }
    __syncthreads();


    if (uplo == MagmaUpper)
    {
        for (int i = 0; i < n; i += BLOCK_SIZE)
        {
            int jb = min(BLOCK_SIZE, n - i);
            col -= jb;

            gemvn_template_device<float, BLK_X, BLK_Y, TILE_SIZE>(jb, i, MAGMA_S_ONE, A(col, col + jb), lda, sx + col + jb, 1, MAGMA_S_ONE, sx + col, 1);
            __syncthreads();

            strsv_backwards_tri_device<trans, diag>(jb, A(col, col), lda, b + col, incb, sx + col);
            __syncthreads();
        }
    }
    else
    {
        for (int i = 0; i < n; i += BLOCK_SIZE)
        {
            int jb = min(BLOCK_SIZE, n - i);
            col = i;

            gemvn_template_device<float, BLK_X, BLK_Y, TILE_SIZE>(jb, i, MAGMA_S_ONE, A(col, 0), lda, sx, 1, MAGMA_S_ONE, sx + col, 1);
            __syncthreads();

            strsv_forwards_tri_device<trans, diag>(jb, A(col, col), lda, b + col, incb, sx + col);
            __syncthreads();
        }
    }


    for (int j = tx; j < n; j += BLOCK_SIZE)
    {
        x[j] = sx[j]; // write to x in reverse order
    }
    __syncthreads();
}


/******************************************************************************/
template< const int BLOCK_SIZE, const int DIM_X, const int DIM_Y,  const int TILE_SIZE, const int flag, const magma_uplo_t uplo, const magma_trans_t trans, const magma_diag_t diag>
__global__ void
strsv_notrans_kernel_outplace_batched(
    int n,
    float **A_array, int lda,
    float **b_array, int incb,
    float **x_array)
{
    int batchid = blockIdx.z;

    strsv_notrans_device<BLOCK_SIZE, DIM_X, DIM_Y, TILE_SIZE, flag, uplo, trans, diag>(n, A_array[batchid], lda, b_array[batchid], incb, x_array[batchid]);
}


/******************************************************************************/
/*template<const int BLOCK_SIZE, const int DIM_X, const int DIM_Y,  const int TILE_SIZE, const int flag, const magma_uplo_t uplo, const magma_trans_t trans, const magma_diag_t diag>
__global__ void
strsv_trans_kernel_outplace_batched(
    int n,
    float **A_array, int lda,
    float **b_array, int incb,
    float **x_array)
{
    int batchid = blockIdx.z;
    strsv_trans_device<BLOCK_SIZE, DIM_X, DIM_Y, TILE_SIZE, flag, uplo, trans, diag>(n, A_array[batchid], lda, b_array[batchid], incb, x_array[batchid]);
}*/



/******************************************************************************/
extern "C" void
magmablas_strsv_outofplace_batched(
    magma_uplo_t uplo, magma_trans_t trans, magma_diag_t diag,
    magma_int_t n,
    float ** A_array, magma_int_t lda,
    float **b_array, magma_int_t incb,
    float **x_array,
    magma_int_t batchCount, hipStream_t queue,
    magma_int_t flag)
{
    /* Check arguments */
    magma_int_t info = 0;
    if ( uplo != MagmaUpper && uplo != MagmaLower ) {
        info = -1;
    } else if ( trans != MagmaNoTrans && trans != MagmaTrans && trans != MagmaConjTrans ) {
        info = -2;
    } else if ( diag != MagmaUnit && diag != MagmaNonUnit ) {
        info = -3;
    } else if (n < 0) {
        info = -5;
    } else if (lda < max(1,n)) {
        info = -8;
    }

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;
    }

    
    // quick return if possible.
    if (n == 0)
        return;

    dim3 threads( NUM_THREADS, 1, 1 );
    dim3 blocks( 1, 1, batchCount );
    size_t shmem = n * sizeof(float);

    if (trans == MagmaNoTrans)
    {
        if (uplo == MagmaUpper)
        {
            if (diag == MagmaNonUnit)
            {
                if (flag == 0) {
                    strsv_notrans_kernel_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 0, MagmaUpper, MagmaNoTrans, MagmaNonUnit >
                        <<< blocks, threads, shmem, queue  >>>
                        (n, A_array, lda, b_array, incb, x_array);
                }
                else {
                    strsv_notrans_kernel_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 1, MagmaUpper, MagmaNoTrans, MagmaNonUnit >
                        <<< blocks, threads, shmem, queue  >>>
                        (n, A_array, lda, b_array, incb, x_array);
                }
            }
            else if (diag == MagmaUnit)
            {
                if (flag == 0) {
                    strsv_notrans_kernel_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 0, MagmaUpper, MagmaNoTrans, MagmaUnit >
                        <<< blocks, threads, shmem, queue  >>>
                        (n, A_array, lda, b_array, incb, x_array);
                }
                else {
                    strsv_notrans_kernel_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 1, MagmaUpper, MagmaNoTrans, MagmaUnit >
                        <<< blocks, threads, shmem, queue  >>>
                        (n, A_array, lda, b_array, incb, x_array);
                }
            }
        }
        else //Lower
        {
            if (diag == MagmaNonUnit)
            {
                if (flag == 0)
                {
                    strsv_notrans_kernel_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 0, MagmaLower, MagmaNoTrans, MagmaNonUnit >
                        <<< blocks, threads, shmem, queue  >>>
                        (n, A_array, lda, b_array, incb, x_array);
                }
                else {
                    strsv_notrans_kernel_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 1, MagmaLower, MagmaNoTrans, MagmaNonUnit >
                        <<< blocks, threads, shmem, queue  >>>
                        (n, A_array, lda, b_array, incb, x_array);
                }
            }
            else if (diag == MagmaUnit)
            {
                if (flag == 0)
                {
                    strsv_notrans_kernel_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 0, MagmaLower, MagmaNoTrans, MagmaUnit>
                        <<< blocks, threads, shmem, queue  >>>
                        (n, A_array, lda, b_array, incb, x_array);
                }
                else {
                    strsv_notrans_kernel_outplace_batched< BLOCK_SIZE_N, DIM_X_N, DIM_Y_N, MagmaBigTileSize, 1, MagmaLower, MagmaNoTrans, MagmaUnit>
                        <<< blocks, threads, shmem, queue  >>>
                        (n, A_array, lda, b_array, incb, x_array);
                }
            }
        }
    }
    else if (trans == MagmaTrans)
    {
        printf("Unhandled code path in magmablas_strsv_outofplace_batched(): trans= MagmaTrans\n");
    }
    else if (trans == MagmaConjTrans)
    {
        printf("Unhandled code path in magmablas_strsv_outofplace_batched(): trans= MagmaConjTrans\n");       
    }
}
